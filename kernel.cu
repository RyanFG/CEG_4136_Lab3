#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

// Dimensions des matrices
#define N 256  // Taille des matrices (N x N)
#define M 16

// Kernel CUDA pour la multiplication de matrices
__global__ void matrixMultiply(float* A, float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float cols[M * N]; // 4096 Memory Spaces?
    __shared__ float rows[M * N]; // 4096 Memory Spaces?

    for (int i = 0; i < N/M; i++) {
        rows[threadIdx.y * M + (N / M) * threadIdx.x + i] = A[row * N + (N / M) * threadIdx.x + i];
        cols[threadIdx.x * M + (N / M) * threadIdx.y + i] = B[col + ((N/M)*threadIdx.y + i)*N];
    }
    __syncthreads();

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += rows[M*threadIdx.y+k]*cols[M*threadIdx.x + k];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);

    // Allouer la mémoire sur le CPU (hôte)
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialiser les matrices A et B avec des valeurs
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // Allouer la mémoire sur le GPU (périphérique)
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copier les matrices A et B du CPU vers le GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Définir la taille des blocs et des grilles
    dim3 block(M, M);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    // Lancer le kernel pour la multiplication de matrices
    matrixMultiply << <grid, block >> > (d_A, d_B, d_C);

    // Attendre la fin de l'exécution du kernel
    hipDeviceSynchronize();

    // Copier le résultat du GPU vers le CPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Afficher un élément du résultat pour vérifier
    for (int i = 0; i < N * N; i++) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    // Libérer la mémoire
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}