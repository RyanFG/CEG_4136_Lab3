#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

// Dimensions des matrices
#define N 3  // Taille des matrices (N x N)

// Kernel CUDA pour la multiplication de matrices
__global__ void matrixMultiply(float* A, float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);

    // Allouer la mémoire sur le CPU (hôte)
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialiser les matrices A et B avec des valeurs
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // Allouer la mémoire sur le GPU (périphérique)
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copier les matrices A et B du CPU vers le GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Définir la taille des blocs et des grilles
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    // Lancer le kernel pour la multiplication de matrices
    matrixMultiply << <grid, block >> > (d_A, d_B, d_C);

    // Attendre la fin de l'exécution du kernel
    hipDeviceSynchronize();

    // Copier le résultat du GPU vers le CPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Afficher un élément du résultat pour vérifier
    for (int i = 0; i < N * N; i++) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    // Libérer la mémoire
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}